#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024

__global__ void matmul(float *A, float *B, float *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    int n = N;
    size_t size = n * n * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    for (int i = 0; i < n*n; i++) {
        h_A[i] = (i % 100) / 100.0f;
        h_B[i] = ((i*2) % 100) / 100.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((n + 15) / 16, (n + 15) / 16);

    matmul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    double checksum = 0.0;
    for (int i = 0; i < n*n; i++) checksum += h_C[i];
    printf("Checksum = %f\n", checksum);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
