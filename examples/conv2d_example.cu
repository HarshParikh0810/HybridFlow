#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 32
#define K 3

__global__ void conv2d(float *input, float *kernel, float *output) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N - K + 1 && j < N - K + 1) {
        float sum = 0.0f;
        for (int ki = 0; ki < K; ki++) {
            for (int kj = 0; kj < K; kj++) {
                sum += input[(i+ki)*N + (j+kj)] * kernel[ki*K + kj];
            }
        }
        output[i*(N-K+1) + j] = sum;
    }
}

int main() {
    int out_size = (N-K+1)*(N-K+1);
    float *h_in = new float[N*N]();
    float *h_kernel = new float[K*K]();
    float *h_out = new float[out_size]();

    float *d_in, *d_kernel, *d_out;
    hipMalloc(&d_in, N*N*sizeof(float));
    hipMalloc(&d_kernel, K*K*sizeof(float));
    hipMalloc(&d_out, out_size*sizeof(float));

    hipMemcpy(d_in, h_in, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernel, K*K*sizeof(float), hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((N-K+1+15)/16, (N-K+1+15)/16);
    conv2d<<<blocks, threads>>>(d_in, d_kernel, d_out);

    hipMemcpy(h_out, d_out, out_size*sizeof(float), hipMemcpyDeviceToHost);

    printf("GPU conv2d complete\n");

    hipFree(d_in); hipFree(d_kernel); hipFree(d_out);
    delete[] h_in; delete[] h_kernel; delete[] h_out;

    return 0;
}
